#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <vector>

using namespace std;

__global__ void addElementsToBucket (int* in, int* s, int *count, int *b, int *size_in) {
    //extern __shared__ int countLocal[];
    int index = threadIdx.x;
    // Binary Searching to find the corresponding bucket.
    int high = *b, low = 0;
    while (high - low > 1) {
        int mid = low + (high - low) / 2;
        if (s[mid] <= in[index]) {
            low = mid;
        }
        else high = mid;
    }
    //printf("%d %d %d \n", index, low, count[low]);
    atomicAdd(&count[low], 1);
    //printf("%d %d %d \n\n", index, low, count[low]);

    //__syncthreads();
    //if (index == *size_in) {
        //for (int i = 0; i < *b; ++i) {
          //  count[i] = countLocal[i];
       // }
    //}
}

__global__ void filterElements (int* in, int* left, int* right, int* out, int *i) {
    int index = threadIdx.x;
    if (in[index] >= *left && in[index] < *right) {
        // To synchronize addition
        int old_i = atomicAdd(i, 1);
        out[old_i] = in[index];
    }
}

vector<int> getSplitterElements (vector<int> a, int b) {
    // Function to generate splitter elements from the original array.
    // log(n) splitter elements will be generated.
    vector<int> s(b + 1);
    s[0] = INT_MIN;
    s[b] = INT_MAX;
    for (int i = 1; i < b; ++i) {
        int index = rand() % a.size();
        s[i] = a[index];
        swap(a[index], a[a.size() - 1]);
        a.pop_back();
    }
    return s;
}

int main() {
    int N = 15;
    vector<int> a(N);
    srand(time(0));
    generate(begin(a), end(a), []() { return rand(); });
    for (int x : a) cout << x << " ";
    cout << endl;
    int k = rand() % (N - 1) + 1;
    cout << k << endl;
   // vector<int> b = a;
   // sort(b.begin(), b.end());
   // cout << b[k - 1] << endl;
    while (a.size() > 4) {
        int b = ceil(log2(a.size())) + 1;
        vector<int> s = getSplitterElements(a, b);
        sort(s.begin(), s.end());
        vector<int> count(b, 0); // Counting the number of elements in each bucket.
        int * d_a, * d_s, * d_count, * d_b, * d_sz;
        int x = a.size();
        hipMalloc(&d_a, a.size() * sizeof(int));
        hipMalloc(&d_s, s.size() * sizeof(int));
        hipMalloc(&d_count, count.size() * sizeof(int));
        hipMalloc(&d_b, sizeof(int));
        hipMalloc(&d_sz, sizeof(int));
        hipMemcpy(d_a, a.data(), a.size() * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_s, s.data(), s.size() * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_count, count.data(), count.size() * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_sz, &x, sizeof(int), hipMemcpyHostToDevice);
        addElementsToBucket << <1, a.size()>> > (d_a, d_s, d_count, d_b, d_sz);
        hipMemcpy(count.data(), d_count, count.size() * sizeof(int), hipMemcpyDeviceToHost);
        int choosenBucket, numElements;
        int l, r;
        // Identifying the current bucket
        for (int i = 0; i < count.size(); ++i) {
            // prefix sums
            if (i) count[i] += count[i - 1];
            if (count[i] >= k) {
                // count[i - 1] < k
                choosenBucket = i;
                if (i) k -= count[i - 1];
                numElements = count[i] - ((i != 0) ? count[i - 1] : 0);
                l = s[i];
                r = s[i + 1];
                break;
            }
        }
        int* d_elementInBucket, * d_l, * d_r, *i;
        int y = 0;
        hipMalloc(&d_elementInBucket, numElements * sizeof(int));
        hipMalloc(&d_l, sizeof(int));
        hipMalloc(&d_r, sizeof(int));
        hipMalloc(&i, sizeof(int));
        hipMemcpy(d_l, &l, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_r, &r, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(i, &y, sizeof(int), hipMemcpyHostToDevice);
        filterElements << <1, a.size() >> > (d_a, d_l, d_r, d_elementInBucket, i);
        a.resize(numElements, 0);
        hipMemcpy(a.data(), d_elementInBucket, numElements * sizeof(int), hipMemcpyDeviceToHost);
      //  break;
    }
    sort(a.begin(), a.end());
    cout << a[k - 1] << endl;
    return 0;
}